#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <time.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>

#define SLICE_LEN 100000000
#define MOBILE_LEN 11
#define BLOCK_LEN 64 // In bytes
#define STATE_LEN 4  // In words
#define LENGTH_SIZE 8 // In bytes
#define HASH_LEN 32

typedef struct
{
    uint8_t mobile[BLOCK_LEN];
    uint32_t hash[STATE_LEN];
} MobileHash;

typedef struct
{
    int index;
    int index_dup;
    MobileHash mobile_hash;
} SortedMobileHash;

typedef char HashString[HASH_LEN];
typedef struct
{
    HashString* hash_string;
    size_t hash_len;
    MobileHash* mobile_hash;
    SortedMobileHash* s_mobile_hash;
    size_t dedup_len;
    size_t count;
} Decoder;

// constants
static const uint8_t PREFIX_LIST[] =
{
186, 158, 135, 159,
136, 150, 137, 138,
187, 151, 182, 152,
139, 183, 188, 134,
185, 189, 180, 157,
155, 156, 131, 132,
133, 130, 181, 176,
177, 153, 184, 178,
173, 147, 175, 199,
166, 170, 198, 171,
191, 145, 165, 172,
154, 146
};
static const size_t PREFIX_SIZE = sizeof(PREFIX_LIST) / sizeof(PREFIX_LIST[0]);


__device__ void md5_compress(uint32_t state[4], const uint8_t block[64]) {
#define LOADSCHEDULE(i)  \
		schedule[i] = (uint32_t)block[i * 4 + 0] <<  0  \
		            | (uint32_t)block[i * 4 + 1] <<  8  \
		            | (uint32_t)block[i * 4 + 2] << 16  \
		            | (uint32_t)block[i * 4 + 3] << 24;

	uint32_t schedule[16];
	LOADSCHEDULE(0)
		LOADSCHEDULE(1)
		LOADSCHEDULE(2)
		LOADSCHEDULE(3)
		LOADSCHEDULE(4)
		LOADSCHEDULE(5)
		LOADSCHEDULE(6)
		LOADSCHEDULE(7)
		LOADSCHEDULE(8)
		LOADSCHEDULE(9)
		LOADSCHEDULE(10)
		LOADSCHEDULE(11)
		LOADSCHEDULE(12)
		LOADSCHEDULE(13)
		LOADSCHEDULE(14)
		LOADSCHEDULE(15)

#define ROTL32(x, n)  (((0U + (x)) << (n)) | ((x) >> (32 - (n))))  // Assumes that x is uint32_t and 0 < n < 32
#define ROUND0(a, b, c, d, k, s, t)  ROUND_TAIL(a, b, d ^ (b & (c ^ d)), k, s, t)
#define ROUND1(a, b, c, d, k, s, t)  ROUND_TAIL(a, b, c ^ (d & (b ^ c)), k, s, t)
#define ROUND2(a, b, c, d, k, s, t)  ROUND_TAIL(a, b, b ^ c ^ d        , k, s, t)
#define ROUND3(a, b, c, d, k, s, t)  ROUND_TAIL(a, b, c ^ (b | ~d)     , k, s, t)
#define ROUND_TAIL(a, b, expr, k, s, t)    \
		a = 0U + a + (expr) + UINT32_C(t) + schedule[k];  \
		a = 0U + b + ROTL32(a, s);

		uint32_t a = state[0];
	uint32_t b = state[1];
	uint32_t c = state[2];
	uint32_t d = state[3];

	ROUND0(a, b, c, d, 0, 7, 0xD76AA478)
		ROUND0(d, a, b, c, 1, 12, 0xE8C7B756)
		ROUND0(c, d, a, b, 2, 17, 0x242070DB)
		ROUND0(b, c, d, a, 3, 22, 0xC1BDCEEE)
		ROUND0(a, b, c, d, 4, 7, 0xF57C0FAF)
		ROUND0(d, a, b, c, 5, 12, 0x4787C62A)
		ROUND0(c, d, a, b, 6, 17, 0xA8304613)
		ROUND0(b, c, d, a, 7, 22, 0xFD469501)
		ROUND0(a, b, c, d, 8, 7, 0x698098D8)
		ROUND0(d, a, b, c, 9, 12, 0x8B44F7AF)
		ROUND0(c, d, a, b, 10, 17, 0xFFFF5BB1)
		ROUND0(b, c, d, a, 11, 22, 0x895CD7BE)
		ROUND0(a, b, c, d, 12, 7, 0x6B901122)
		ROUND0(d, a, b, c, 13, 12, 0xFD987193)
		ROUND0(c, d, a, b, 14, 17, 0xA679438E)
		ROUND0(b, c, d, a, 15, 22, 0x49B40821)
		ROUND1(a, b, c, d, 1, 5, 0xF61E2562)
		ROUND1(d, a, b, c, 6, 9, 0xC040B340)
		ROUND1(c, d, a, b, 11, 14, 0x265E5A51)
		ROUND1(b, c, d, a, 0, 20, 0xE9B6C7AA)
		ROUND1(a, b, c, d, 5, 5, 0xD62F105D)
		ROUND1(d, a, b, c, 10, 9, 0x02441453)
		ROUND1(c, d, a, b, 15, 14, 0xD8A1E681)
		ROUND1(b, c, d, a, 4, 20, 0xE7D3FBC8)
		ROUND1(a, b, c, d, 9, 5, 0x21E1CDE6)
		ROUND1(d, a, b, c, 14, 9, 0xC33707D6)
		ROUND1(c, d, a, b, 3, 14, 0xF4D50D87)
		ROUND1(b, c, d, a, 8, 20, 0x455A14ED)
		ROUND1(a, b, c, d, 13, 5, 0xA9E3E905)
		ROUND1(d, a, b, c, 2, 9, 0xFCEFA3F8)
		ROUND1(c, d, a, b, 7, 14, 0x676F02D9)
		ROUND1(b, c, d, a, 12, 20, 0x8D2A4C8A)
		ROUND2(a, b, c, d, 5, 4, 0xFFFA3942)
		ROUND2(d, a, b, c, 8, 11, 0x8771F681)
		ROUND2(c, d, a, b, 11, 16, 0x6D9D6122)
		ROUND2(b, c, d, a, 14, 23, 0xFDE5380C)
		ROUND2(a, b, c, d, 1, 4, 0xA4BEEA44)
		ROUND2(d, a, b, c, 4, 11, 0x4BDECFA9)
		ROUND2(c, d, a, b, 7, 16, 0xF6BB4B60)
		ROUND2(b, c, d, a, 10, 23, 0xBEBFBC70)
		ROUND2(a, b, c, d, 13, 4, 0x289B7EC6)
		ROUND2(d, a, b, c, 0, 11, 0xEAA127FA)
		ROUND2(c, d, a, b, 3, 16, 0xD4EF3085)
		ROUND2(b, c, d, a, 6, 23, 0x04881D05)
		ROUND2(a, b, c, d, 9, 4, 0xD9D4D039)
		ROUND2(d, a, b, c, 12, 11, 0xE6DB99E5)
		ROUND2(c, d, a, b, 15, 16, 0x1FA27CF8)
		ROUND2(b, c, d, a, 2, 23, 0xC4AC5665)
		ROUND3(a, b, c, d, 0, 6, 0xF4292244)
		ROUND3(d, a, b, c, 7, 10, 0x432AFF97)
		ROUND3(c, d, a, b, 14, 15, 0xAB9423A7)
		ROUND3(b, c, d, a, 5, 21, 0xFC93A039)
		ROUND3(a, b, c, d, 12, 6, 0x655B59C3)
		ROUND3(d, a, b, c, 3, 10, 0x8F0CCC92)
		ROUND3(c, d, a, b, 10, 15, 0xFFEFF47D)
		ROUND3(b, c, d, a, 1, 21, 0x85845DD1)
		ROUND3(a, b, c, d, 8, 6, 0x6FA87E4F)
		ROUND3(d, a, b, c, 15, 10, 0xFE2CE6E0)
		ROUND3(c, d, a, b, 6, 15, 0xA3014314)
		ROUND3(b, c, d, a, 13, 21, 0x4E0811A1)
		ROUND3(a, b, c, d, 4, 6, 0xF7537E82)
		ROUND3(d, a, b, c, 11, 10, 0xBD3AF235)
		ROUND3(c, d, a, b, 2, 15, 0x2AD7D2BB)
		ROUND3(b, c, d, a, 9, 21, 0xEB86D391)

		state[0] = 0U + state[0] + a;
	state[1] = 0U + state[1] + b;
	state[2] = 0U + state[2] + c;
	state[3] = 0U + state[3] + d;
}

__host__ __device__ int is_equal(const MobileHash* a, const MobileHash* b)
{
    if (a->hash[0] == b->hash[0]
        && a->hash[1] == b->hash[1]
        && a->hash[2] == b->hash[2]
        && a->hash[3] == b->hash[3])
        return 1;

    return 0;
}

__host__ __device__ int is_lesser(const MobileHash* a, const MobileHash* b)
{
    if (a->hash[0] < b->hash[0])
    {
        return 1;
    }
    else if (a->hash[0] == b->hash[0]) {
        if (a->hash[1] < b->hash[1])
        {
            return 1;
        }
        else if (a->hash[1] == b->hash[1]) {
            if (a->hash[2] < b->hash[2])
            {
                return 1;
            }
            else if (a->hash[2] == b->hash[2]) {
                if (a->hash[3] < b->hash[3])
                {
                    return 1;
                }
            }
        }
    }

    return 0;
}

__device__ int binary_search(SortedMobileHash* array, int len, const MobileHash* key)
{
    int low = 0, high = len - 1, mid;
    while (low <= high)
    {
        mid = (low + high) / 2;
        if (is_equal(&array[mid].mobile_hash, key))
        {
            return mid;
        }
        else if (is_lesser(&array[mid].mobile_hash, key))
        {
            low = mid + 1;
        }
        else
        {
            high = mid - 1;
        }
    }
    return -1;
}

__global__ void compute(SortedMobileHash* smh, int dedup_len, uint8_t prefix0, uint8_t prefix1, uint8_t prefix2, char* p_numbers)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < SLICE_LEN)
    {
        MobileHash mh;
        memset(mh.mobile, 0, BLOCK_LEN);
        mh.mobile[0] = prefix0;
        mh.mobile[1] = prefix1;
        mh.mobile[2] = prefix2;
        memcpy(mh.mobile + 3, p_numbers + (i/10000) * 5, 4);
        memcpy(mh.mobile + 7, p_numbers + (i % 10000) * 5, 4);
        mh.mobile[MOBILE_LEN] = 0x80;
        mh.mobile[BLOCK_LEN - LENGTH_SIZE] = 'X';

        mh.hash[0] = UINT32_C(0x67452301);
        mh.hash[1] = UINT32_C(0xEFCDAB89);
        mh.hash[2] = UINT32_C(0x98BADCFE);
        mh.hash[3] = UINT32_C(0x10325476);
        md5_compress(mh.hash, mh.mobile);

        int index = binary_search(smh, dedup_len, &mh);
        if (index >= 0)
        {
            smh[index].mobile_hash = mh;
        }
    }
}


char* read_from_file(char* filename)
{
    FILE* f = fopen(filename, "rb");
    if (f)
    {
        fseek(f, 0, SEEK_END);
        long len = ftell(f);
        fseek(f, 0, SEEK_SET);
        char* buffer = (char*)malloc(len + 1);
        len = fread(buffer, 1, len, f);
        fclose(f);
        buffer[len] = 0;
        return buffer;
    }
    return 0;
}

void write_to_file(char* filename, Decoder* decoder)
{
    FILE* f = fopen(filename, "w");
    for (int h = 0; h < decoder->hash_len; h++)
    {
        for (size_t i = 0; i < HASH_LEN; i++)
        {
            fputc(decoder->hash_string[h][i], f);
        }
        fputc(',', f);
        for (size_t i = 0; i < MOBILE_LEN; i++)
        {
            fputc(decoder->mobile_hash[h].mobile[i], f);
        }
        fputc('\n', f);
    }
    fclose(f);
}

void setupCuda()
{
    hipDeviceProp_t deviceProps;
    if (hipGetDeviceProperties(&deviceProps, 0))
    {
        printf("no CUDA devices\n");
        exit(1);
    }
    printf("use CUDA device [%s]\n", deviceProps.name);
    hipSetDevice(0);
}

void quick_sort(SortedMobileHash* array, int from, int to)
{
    if (from >= to)return;
    SortedMobileHash temp;
    int i = from, j;
    for (j = from + 1; j <= to; j++)
    {
        if (is_lesser(&array[j].mobile_hash, &array[from].mobile_hash))
        {
            i = i + 1;
            temp = array[i];
            array[i] = array[j];
            array[j] = temp;
        }
    }

    temp = array[i];
    array[i] = array[from];
    array[from] = temp;
    quick_sort(array, from, i - 1);
    quick_sort(array, i + 1, to);
}

inline char hexToNibble(char n)
{
    return n - (n <= '9' ? '0' : ('a' - 10));
}
void hex_to_bytes(uint8_t* to, char* from, int len)
{
    for (int i = 0; i < len / 2; i++)
    {
        to[i] = (hexToNibble(from[i * 2]) << 4) + hexToNibble(from[i * 2 + 1]);
    }
}

void print_mobile_hash(MobileHash* mh) {
    for (size_t i = 0; i < STATE_LEN; i++)
    {
        printf("%x-", mh->hash[i]);
    }
    printf(",");
    for (size_t i = 0; i < MOBILE_LEN; i++)
    {
        printf("%c", mh->mobile[i]);
    }
    printf("\n");
}
void print_sorted_mobile_hash(SortedMobileHash* smh) {
    printf("%d,%d,", smh->index, smh->index_dup);
    print_mobile_hash(&smh->mobile_hash);
}

size_t validate_hash_string(const char* s)
{
    int valid_char_num = 0;
    int count = 0;
    while (*s)
    {
        if (*s == ',')
        {
            if (valid_char_num == HASH_LEN)
            {
                count++;
            }
            valid_char_num = 0;
        }
        else if ((*s >= 'a' && *s <= 'z') || (*s >= '0' && *s <= '9'))
        {
            valid_char_num++;
        }
        s++;
    }
    if (valid_char_num == HASH_LEN)
    {
        count++;
    }

    return count;
}

void parse_hash_strings(Decoder& decoder, const char* s)
{
    SortedMobileHash* p_smh = decoder.s_mobile_hash;
    HashString* p_hs = decoder.hash_string;

    int valid_char_num = 0;
    int count = 0;
    char hash_string[HASH_LEN];
    while (*s)
    {
        if (*s == ',')
        {
            if (valid_char_num == HASH_LEN)
            {
                memcpy(p_hs, hash_string, HASH_LEN);
                hex_to_bytes((uint8_t*)p_smh->mobile_hash.hash, hash_string, HASH_LEN);
                p_smh->index = count;
                count++;
                p_smh++;
                p_hs++;
            }
            valid_char_num = 0;
        }
        else if ((*s >= 'a' && *s <= 'z') || (*s >= '0' && *s <= '9'))
        {
            hash_string[valid_char_num] = *s;
            valid_char_num++;
        }
        s++;
    }
    if (valid_char_num == HASH_LEN)
    {
        memcpy(p_hs, hash_string, HASH_LEN);
        hex_to_bytes((uint8_t*)p_smh->mobile_hash.hash, hash_string, HASH_LEN);
        p_smh->index = count;
    }
}

void dedup_sorted_mobile_hash(Decoder& decoder)
{
    decoder.dedup_len = decoder.hash_len;
    for (size_t i = 1; i < decoder.dedup_len; i++)
    {
        if (is_equal(&decoder.s_mobile_hash[i].mobile_hash, &decoder.s_mobile_hash[i - 1].mobile_hash))
        {
            SortedMobileHash temp = decoder.s_mobile_hash[i];
            temp.index_dup = decoder.s_mobile_hash[i - 1].index;
            for (size_t j = i; j < decoder.hash_len - 1; j++)
            {
                decoder.s_mobile_hash[j] = decoder.s_mobile_hash[j + 1];
            }
            decoder.s_mobile_hash[decoder.hash_len - 1] = temp;
            decoder.dedup_len--;
            i--;
        }
    }
}


void resort_mobile_hash(Decoder& decoder)
{
    for (int i = 0; i < decoder.hash_len; i++)
    {
        if (i < decoder.dedup_len)
        {
            int index = decoder.s_mobile_hash[i].index;
            decoder.mobile_hash[index] = decoder.s_mobile_hash[i].mobile_hash;
        }
        else
        {
            int index = decoder.s_mobile_hash[i].index;
            int index_dup = decoder.s_mobile_hash[i].index_dup;
            decoder.mobile_hash[index] = decoder.mobile_hash[index_dup];
        }
    }
}

void init_decoder(Decoder& decoder,const char* s)
{
    decoder.hash_len = validate_hash_string(s);
    decoder.hash_string = (HashString*)calloc(decoder.hash_len, sizeof(HashString));
    decoder.mobile_hash = (MobileHash*)calloc(decoder.hash_len, sizeof(MobileHash));
    decoder.s_mobile_hash = (SortedMobileHash*)calloc(decoder.hash_len, sizeof(SortedMobileHash));
    decoder.count = 0;
    parse_hash_strings(decoder, s);
    quick_sort(decoder.s_mobile_hash, 0, decoder.hash_len - 1);
    dedup_sorted_mobile_hash(decoder);
    // for (size_t i = 0; i < decoder.hash_len; i++)
    // {
    //     printf("%zu,", i);
    //     print_sorted_mobile_hash(decoder.s_mobile_hash+i);
    // }
}
void free_decoder(Decoder& decoder)
{
    free(decoder.hash_string);
    decoder.hash_string = 0;
    free(decoder.mobile_hash);
    decoder.mobile_hash = 0;
    free(decoder.s_mobile_hash);
    decoder.s_mobile_hash = 0;
}

void check(hipError_t e, int const line)
{
    if (e)
    {
        printf("hipError_t(%d) @%d\n", e, line);
        exit(e);
    }
}

#define check_error(val) check(val, __LINE__)

int main(int argc, char* argv[])
{
    time_t start = time(NULL);

    // setup cuda
    setupCuda();

    // process hash file
    char* s = NULL;
    if (argc < 2 || !(s = read_from_file(argv[1])))
    {
        printf("mobile md5 decoder [cuda], v1.0, by herman\n");
        printf("usage: mm filename\n");
        return 1;
    }

    Decoder decoder;
    init_decoder(decoder, s);
    free(s);
    s = 0;
    printf("find %zu hashes\n", decoder.hash_len);
    printf("they have %zu duplicated, %zu unique ones\n", decoder.hash_len - decoder.dedup_len, decoder.dedup_len);

    // compute helper strings
    char* p_numbers = (char *)malloc(10000 * 5);
    for (size_t i = 0; i < 10000; i++)
    {
        sprintf(p_numbers+i*5, "%04d", i);
    }

    // allocate device memory
    SortedMobileHash* d_smh;
    size_t smh_len = decoder.dedup_len * sizeof(SortedMobileHash);
    check_error(hipMalloc((void**)&d_smh, smh_len));
    check_error(hipMemcpy(d_smh, decoder.s_mobile_hash, smh_len, hipMemcpyHostToDevice));
    char* d_p_numbers;
    check_error(hipMalloc((void**)&d_p_numbers, 10000 * 5));
    check_error(hipMemcpy(d_p_numbers, p_numbers, 10000 * 5, hipMemcpyHostToDevice));

    // release host memory
    free(p_numbers);

    // set kernel launch configuration
    int threads = 512;
    int blocks = (SLICE_LEN+511) / 512;

    printf("0%% @%lds - 0/%zu\n", time(NULL) - start, decoder.dedup_len);
    // work on each prefix
    for (size_t i = 0; i < PREFIX_SIZE; i++)
    {
        uint8_t prefix[3];
        prefix[0] = PREFIX_LIST[i] / 100 + '0';
        prefix[1] = (PREFIX_LIST[i] % 100) / 10 + '0';
        prefix[2] = PREFIX_LIST[i] % 10 + '0';
        compute << <blocks, threads >> > (d_smh, decoder.dedup_len, prefix[0], prefix[1], prefix[2], d_p_numbers);
        check_error(hipDeviceSynchronize());
        check_error(hipMemcpy(decoder.s_mobile_hash, d_smh, smh_len, hipMemcpyDeviceToHost));
        decoder.count = 0;
        for (size_t h = 0; h < decoder.dedup_len; h++)
        {
            if (decoder.s_mobile_hash[h].mobile_hash.mobile[0])
            {
                decoder.count++;
            }
        }
        printf("\033[1A%d%% @%lds - %zu/%zu\n", (i+1)*100/ PREFIX_SIZE, time(NULL) - start, decoder.count, decoder.dedup_len);
        if (decoder.count == decoder.dedup_len)
        {
            break;
        }
    }

    // write reults
    printf("total %zu hashes are decoded\n", decoder.count);
    resort_mobile_hash(decoder);
    size_t fn_len = strlen(argv[1]) + 5;
    char* outfile = (char*)malloc(fn_len);
    strcpy(outfile, argv[1]);
    strcat(outfile, ".out");
    write_to_file(outfile, &decoder);
    printf("please find results in file: %s\n", outfile);
    free(outfile);

    // release resources
    hipFree(d_smh);
    hipFree(d_p_numbers);
    free_decoder(decoder);
}
